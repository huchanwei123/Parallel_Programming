#include "hip/hip_runtime.h"
/****************************************************************************************
    All-paired shortest path implementation in CUDA (Multi-Node version)
    Optimization:
        1. Unroll
        2. shared memory in phase 3
    Author:
        Chan-Wei Hu
******************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <mpi.h>

#define inf 1e9
static int block_dim = 32;

// phase 1 kernel (done!!)
__global__ void Phase_1(int *adj_mat_d, int round, int block_dim, int comp_V) {
    
	int i = threadIdx.y, 
        j = threadIdx.x,
        offset = block_dim * round;
    
    extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + offset) * comp_V + (j + offset)];
	__syncthreads();

#pragma unroll
	for(int k = 0; k < block_dim; k++){
        if (shared_mem[i * block_dim + j] > shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j]){
            shared_mem[i * block_dim + j] = shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j];
        }
        __syncthreads();
	}
	adj_mat_d[(i + offset) * comp_V + (j + offset)] = shared_mem[i * block_dim + j];
    __syncthreads();
}

// phase 2 kernel (done !!!)
__global__ void Phase_2(int* adj_mat_d, int round, int block_dim, int comp_V) {
	
    int total_round = comp_V/block_dim;

    int i = threadIdx.y,
        j = threadIdx.x,
        // column or row?
        i_off = blockIdx.x == 1? block_dim * ((blockIdx.y + round + 1) % total_round): block_dim * round,
        j_off = blockIdx.x == 1? block_dim * round : block_dim * ((blockIdx.y + round + 1) % total_round);
	
    extern __shared__ int shared_mem[];
	
    shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
	__syncthreads();

#pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]) {
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]; 
            
            if (round == i_off/block_dim) 
                shared_mem[(i + 2*block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
            if (round == j_off/block_dim) 
                shared_mem[(i + block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
		}	
	}
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

// Phase 3 kernel (done !!!)
__global__ void Phase_3(int* adj_mat_d, int round, int block_dim, int comp_V, int offset) {

    int i = threadIdx.y,
        j = threadIdx.x,
        i_off = block_dim * (blockIdx.x + offset),
        j_off = block_dim * blockIdx.y;
     
	extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
    __syncthreads();

#pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j])
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j];
	}
	
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

int main(int argc, char *argv[]){
    // Initial MPI
    int size, rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	/******************************* load data *********************************/
    // only two arguments are allowed
    assert(argc == 3);

    int E, V;
    FILE *in_fp;
    in_fp = fopen(argv[1], "r");
    if(in_fp == NULL) printf("Failed on opening file\n");
    // read in data
    fread(&V, sizeof(int), 1, in_fp);
    fread(&E, sizeof(int), 1, in_fp);

    // compensate V to make V % block_dim == 0
	int comp_V = V + (block_dim - ((V-1) % block_dim + 1));

	//allocate memory
    int *adj_mat; 
    size_t sz = comp_V * comp_V * sizeof(int);
	hipHostMalloc((void**) &adj_mat, sz);
	for(int i = 0; i < comp_V; i++){
        for(int j = 0; j < comp_V; j++){
            if(i == j) adj_mat[i*comp_V+j] = 0;
            else adj_mat[i*comp_V+j] = inf;
        }
    }
    // load data to graph
    int src, dst, w;
    while(E--){
        fread(&src, sizeof(int), 1, in_fp);
        fread(&dst, sizeof(int), 1, in_fp);
        fread(&w, sizeof(int), 1, in_fp);
        adj_mat[src*comp_V+dst] = w;
    }
    fclose(in_fp);
    /****************************************************************************/

    int *adj_mat_d;
	int round =  ceil((float) comp_V/block_dim);
	
    // 2D block
    dim3 threads(block_dim, block_dim);
	
	dim3 p1(1, 1);
	dim3 p2(2, round-1);

    hipSetDevice(rank);

    // Malloc memory
    hipMalloc(&adj_mat_d, sz);

    // divide data
    int round_per_thd = round / 2;
    int y_offset = round_per_thd * rank;
    if(rank == size-1)
        round_per_thd += round % 2;

    dim3 p3(round_per_thd, round);
    
    size_t cp_amount = comp_V * block_dim * round_per_thd * sizeof(int);
    hipMemcpy(adj_mat_d + y_offset *block_dim * comp_V, adj_mat + y_offset * block_dim * comp_V, cp_amount, hipMemcpyHostToDevice);

    size_t block_row_sz = block_dim * comp_V * sizeof(int);
    for(int r = 0; r < round; r++){    
        if (r >= y_offset && r < (y_offset + round_per_thd)) {
            hipMemcpy(adj_mat + r * block_dim * comp_V, adj_mat_d + r * block_dim * comp_V, block_row_sz, hipMemcpyDeviceToHost);
            // send to another node
            MPI_Send(adj_mat + r * block_dim * comp_V, comp_V * block_dim, MPI_INT, (rank + 1) % 2, 0, MPI_COMM_WORLD);
        }else{
            // Another recieve
            MPI_Recv(adj_mat + r * block_dim * comp_V, comp_V * block_dim, MPI_INT, (rank + 1) % 2, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }
        hipMemcpy(adj_mat_d + r * block_dim * comp_V, adj_mat + r * block_dim * comp_V, block_row_sz, hipMemcpyHostToDevice);

        Phase_1 <<<p1, threads, sizeof(int)*block_dim*block_dim >>>(adj_mat_d, r, block_dim, comp_V);
        
        hipDeviceSynchronize();
        
        Phase_2 <<<p2, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d, r, block_dim, comp_V);
        
        hipDeviceSynchronize();
        
        Phase_3 <<<p3, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d, r, comp_V, y_offset);
    }
    hipMemcpy(adj_mat + y_offset *block_dim * comp_V, adj_mat_d + y_offset *block_dim * comp_V, block_row_sz * round_per_thd, hipMemcpyDeviceToHost);

    // send data back to rank 0
    if(rank == 0){
        int offset = round_per_thd * block_dim * comp_V;
        int remain = comp_V * comp_V - offset;
        MPI_Recv(adj_mat + offset, remain, MPI_INT, 1, 0,  MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }else{
        MPI_Send(adj_mat + y_offset * block_dim * comp_V, round_per_thd * block_dim * comp_V, MPI_INT, 0, 0, MPI_COMM_WORLD);
    }
	
	// output
    if(rank == 0){
        FILE *out_fp;
        out_fp = fopen(argv[2], "wb");
        for(int i = 0; i < V; i++){
            for(int j = 0; j < V; j++){
                fwrite(adj_mat+i*comp_V+j, sizeof(int), 1, out_fp);
            }   
        }   
        fclose(out_fp);
    }
	//free memory
	hipFree(adj_mat_d);
    hipHostFree(adj_mat);
	
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();
	return 0;
}
