/****************************************************************************************
    All-paired shortest path implementation in CUDA
    Optimization:
        1. Unroll
        2. shared memory in phase 3
    Author:
        Chan-Wei Hu
******************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <omp.h>

#define inf 1e9
static int block_dim = 32;

// phase 1 kernel (done!!)
__global__ void Phase_1(int *adj_mat_d, int round, int block_dim, int comp_V) {
    
	int i = threadIdx.y, 
        j = threadIdx.x,
        offset = block_dim * round;
    
    extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + offset) * comp_V + (j + offset)];
	__syncthreads();

#pragma unroll
	for(int k = 0; k < block_dim; k++){
        if (shared_mem[i * block_dim + j] > shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j]){
            shared_mem[i * block_dim + j] = shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j];
        }
        __syncthreads();
	}
	adj_mat_d[(i + offset) * comp_V + (j + offset)] = shared_mem[i * block_dim + j];
    __syncthreads();
}

// phase 2 kernel (done !!!)
__global__ void Phase_2(int* adj_mat_d, int round, int block_dim, int comp_V) {
	
    int total_round = comp_V/block_dim;

    int i = threadIdx.y,
        j = threadIdx.x,
        // column or row?
        i_off = blockIdx.x == 1? block_dim * ((blockIdx.y + round + 1) % total_round): block_dim * round,
        j_off = blockIdx.x == 1? block_dim * round : block_dim * ((blockIdx.y + round + 1) % total_round);
	
    extern __shared__ int shared_mem[];
	
    shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
	__syncthreads();

#pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]) {
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]; 
            
            if (round == i_off/block_dim) 
                shared_mem[(i + 2*block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
            if (round == j_off/block_dim) 
                shared_mem[(i + block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
		}	
	}
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

// Phase 3 kernel (done !!!)
__global__ void Phase_3(int* adj_mat_d, int round, int block_dim, int comp_V, int offset) {

    int i = threadIdx.y,
        j = threadIdx.x,
        i_off = block_dim * (blockIdx.x + offset),
        j_off = block_dim * blockIdx.y;

     
	extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
    __syncthreads();
	
#pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j])
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j];
	}
	
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

int main(int argc, char *argv[]){
	/******************************* load data *********************************/
    // only two arguments are allowed
    assert(argc == 3);

    int E, V;
    FILE *in_fp;
    in_fp = fopen(argv[1], "r");
    if(in_fp == NULL) printf("Failed on opening file\n");
    // read in data
    fread(&V, sizeof(int), 1, in_fp);
    fread(&E, sizeof(int), 1, in_fp);

    // compensate V to make V % block_dim == 0
	int comp_V = V + (block_dim - ((V-1) % block_dim + 1));

	//allocate memory
    int *adj_mat; 
    size_t sz = comp_V * comp_V * sizeof(int);
	hipHostMalloc((void**) &adj_mat, sz, hipHostMallocDefault);
	for(int i = 0; i < comp_V; i++){
        for(int j = 0; j < comp_V; j++){
            if(i == j) adj_mat[i*comp_V+j] = 0;
            else adj_mat[i*comp_V+j] = inf;
        }
    }
    // load data to graph
    int src, dst, w;
    while(E--){
        fread(&src, sizeof(int), 1, in_fp);
        fread(&dst, sizeof(int), 1, in_fp);
        fread(&w, sizeof(int), 1, in_fp);
        adj_mat[src*comp_V+dst] = w;
    }
    fclose(in_fp);
    /****************************************************************************/

    int *adj_mat_d[2];
	int round =  ceil((float) comp_V/block_dim);
	
    // 2D block
    dim3 threads(block_dim, block_dim);
	
	dim3 p1(1, 1);
	dim3 p2(2, round-1);

	#pragma omp parallel num_threads(2)
	{
		int thread_id = omp_get_thread_num();
		hipSetDevice(thread_id);

        // Malloc memory
        hipMalloc((void**) &adj_mat_d[thread_id], sz);

        // divide data
		int round_per_thd = round / 2;
		int y_offset = round_per_thd * thread_id;
        if(thread_id == 1)
			round_per_thd += round % 2;

		dim3 p3(round_per_thd, round);
		
	    size_t cp_amount = comp_V * block_dim * round_per_thd * sizeof(int);
        hipMemcpy(adj_mat_d[thread_id] + y_offset *block_dim * comp_V, adj_mat + y_offset * block_dim * comp_V, cp_amount, hipMemcpyHostToDevice);

        size_t block_row_sz = block_dim * comp_V * sizeof(int);
		for(int r = 0; r < round; r++){    
			if (r >= y_offset && r < (y_offset + round_per_thd)) {
				hipMemcpy(adj_mat + r * block_dim * comp_V, adj_mat_d[thread_id] + r * block_dim * comp_V, block_row_sz, hipMemcpyDeviceToHost);
			}
			#pragma omp barrier
			hipMemcpy(adj_mat_d[thread_id] + r * block_dim * comp_V, adj_mat + r * block_dim * comp_V, block_row_sz, hipMemcpyHostToDevice);

			Phase_1 <<<p1, threads, sizeof(int)*block_dim*block_dim >>>(adj_mat_d[thread_id], r, block_dim, comp_V);
            
            hipDeviceSynchronize();
			
            Phase_2 <<<p2, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d[thread_id], r, block_dim, comp_V);
			
            hipDeviceSynchronize();
			
            Phase_3 <<<p3, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d[thread_id], r, block_dim, comp_V, y_offset);
		}
		hipMemcpy(adj_mat + y_offset *block_dim * comp_V, adj_mat_d[thread_id] + y_offset *block_dim * comp_V, block_row_sz * round_per_thd, hipMemcpyDeviceToHost);
		#pragma omp barrier
	}
	
	// output
    FILE *out_fp;
    out_fp = fopen(argv[2], "wb");
    for(int i = 0; i < V; i++){
        for(int j = 0; j < V; j++){
            fwrite(adj_mat+i*comp_V+j, sizeof(int), 1, out_fp);
        }   
    }   
    fclose(out_fp);

	//free memory
	hipFree(adj_mat_d[0]);
    hipFree(adj_mat_d[1]);
    hipHostFree(adj_mat);
	return 0;
}
