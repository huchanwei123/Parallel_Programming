#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

#define inf 1e9

__global__ void update(int *adj_mat_d, int Xstart, int Ystart, int round, int B, int comp_V){
    int k;
    int i = B * Xstart + blockIdx.x * blockDim.x + threadIdx.x;
    int j = B * Ystart + blockIdx.y * blockDim.y + threadIdx.y;
    //if(blockIdx.x >= 11) printf("block id = %d\n", blockIdx.x);
    for(int offset = 0; offset < B; offset++){
        k = round*B+offset;
        if(adj_mat_d[i*comp_V+j] > adj_mat_d[i*comp_V+k] + adj_mat_d[k*comp_V+j]){
            adj_mat_d[i*comp_V+j] = adj_mat_d[i*comp_V+k] + adj_mat_d[k*comp_V+j];
        }
    }
}

int main(int argc, char *argv[]){
    /******************************* load data *********************************/
    // only two arguments are allowed
    assert(argc == 3);

    // open input file
    int E, comp_V, V;
    FILE *in_fp;
    in_fp = fopen(argv[1], "r");
    if(in_fp == NULL) printf("Failed on opening file\n");

    // read in data
    fread(&V, sizeof(int), 1, in_fp);
    fread(&E, sizeof(int), 1, in_fp);
    printf("Total vertices: %d\nTotal edges: %d\n", V, E);

    // start dividing data
    // block size
    int B = 16;
    printf("Block size B = %d\n", B);

    // check if V % B == 0
    int V_block = V % B ? V/B+1 : V/B;
    comp_V = B*V_block+1;
    // create adjacency matrix for new graph
    int *adj_mat = (int*)malloc(comp_V*comp_V*sizeof(int));
    for(int i = 0; i < comp_V; i++){
        for(int j = 0; j < comp_V; j++){
            if(i == j) adj_mat[i*comp_V+j] = 0;
            else adj_mat[i*comp_V+j] = inf;
        }
    }
    
    // load data to graph
    int src, dst, w;
    while(E--){
        fread(&src, sizeof(int), 1, in_fp);
        fread(&dst, sizeof(int), 1, in_fp);
        fread(&w, sizeof(int), 1, in_fp);
        if(adj_mat[src*comp_V+dst] > w) adj_mat[src*comp_V+dst] = w;
    }
    fclose(in_fp);
    /****************************************************************************/
    
    /****************************** Device info. ********************************/
    // only single GPU is used, so the device index is 0
    // print out the information of GPU
    int device_idx = 0;
    /*
    cudaDeviceProp deviceProp;
    cudaError_t cudaError;
    cudaError = cudaGetDeviceProperties(&deviceProp, device_idx);
    printf("Device max threads per block : %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max thread dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    */
    hipSetDevice(device_idx);
    /****************************************************************************/
    // block size
    int round = ceil((float)V/B);
    int block_dim = 8;
    int grid_size = B/block_dim;
    //int gridFactor = 1;
    //int gridFactor = (1024/B)*(1024/B);
    //int gridFactor = B*B;
    printf("Will run %d rounds\n", round);

    dim3 blocks(grid_size, grid_size);
    dim3 threads(block_dim, block_dim);
    dim3 Col_Other(grid_size*round, grid_size);
    dim3 Row_Other(grid_size, grid_size*round);

    // malloc for cuda memory 
    int *adj_mat_d;
    hipMalloc((void**) &adj_mat_d, comp_V*comp_V*sizeof(int));
    hipMemcpy(adj_mat_d, adj_mat, comp_V*comp_V*sizeof(int), hipMemcpyHostToDevice);

    // start iteration
    for(int r = 0; r < round; r++){
        /********************** phase 1 ***********************/
        update <<<blocks, threads>>> (adj_mat_d, r, r, r, B, comp_V);

        /********************** phase 2 ***********************/
        // column part
        update <<<Col_Other, threads>>> (adj_mat_d, 0, r, r, B, comp_V);
        // row part 
        update <<<Row_Other, threads>>> (adj_mat_d, r, 0, r, B, comp_V);
        
        /********************** phase 3 ************************/
        for(int i = 0; i < round; i++){
            update <<<Row_Other, threads>>> (adj_mat_d, i, 0, r, B, comp_V);
        }
    }
    hipDeviceSynchronize();
    // copy result back to host
    hipMemcpy(adj_mat, adj_mat_d, comp_V*comp_V*sizeof(int), hipMemcpyDeviceToHost);
    // free memory
    hipFree(adj_mat_d);

    // output the result
    FILE *out_fp;
    out_fp = fopen(argv[2], "wb");
    for(int i = 0; i < V; i++){
        for(int j = 0; j < V; j++){
            fwrite(adj_mat+i*comp_V+j, sizeof(int), 1, out_fp);
        }   
    }   
    fclose(out_fp);
    hipHostFree(adj_mat);

    return 0;
}

