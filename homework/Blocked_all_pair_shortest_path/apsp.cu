/****************************************************************************************
    All-paired shortest path implementation in CUDA (single GPU version)
    Optimization:
        1. Unroll
        2. shared memory in phase 3
    Author:
        Chan-Wei Hu
******************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <omp.h>

#define inf 1e9
static int block_dim = 32;

// phase 1 kernel (done!!)
__global__ void Phase_1(int *adj_mat_d, int round, int block_dim, int comp_V) {
    
	int i = threadIdx.y, 
        j = threadIdx.x,
        offset = block_dim * round;
    
    extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + offset) * comp_V + (j + offset)];
	__syncthreads();

    #pragma unroll
	for(int k = 0; k < block_dim; k++){
        if (shared_mem[i * block_dim + j] > shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j]){
            shared_mem[i * block_dim + j] = shared_mem[i * block_dim + k] + shared_mem[k * block_dim + j];
        }
        __syncthreads();
	}
	adj_mat_d[(i + offset) * comp_V + (j + offset)] = shared_mem[i * block_dim + j];
    __syncthreads();
}

// phase 2 kernel (done !!!)
__global__ void Phase_2(int* adj_mat_d, int round, int block_dim, int comp_V) {
	
    int total_round = comp_V/block_dim;

    int i = threadIdx.y,
        j = threadIdx.x,
        // column or row?
        i_off = blockIdx.x == 1? block_dim * ((blockIdx.y + round + 1) % total_round): block_dim * round,
        j_off = blockIdx.x == 1? block_dim * round : block_dim * ((blockIdx.y + round + 1) % total_round);
	
    extern __shared__ int shared_mem[];
	
    shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
	__syncthreads();

    #pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]) {
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j]; 
            
            if (round == i_off/block_dim) 
                shared_mem[(i + 2*block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
            if (round == j_off/block_dim) 
                shared_mem[(i + block_dim) * block_dim + j] = shared_mem[i * block_dim + j];
		}	
	}
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

// Phase 3 kernel (done !!!)
__global__ void Phase_3(int* adj_mat_d, int round, int block_dim, int comp_V){

    int i = threadIdx.y,
        j = threadIdx.x,
        i_off = block_dim * blockIdx.x,
        j_off = block_dim * blockIdx.y;

     
	extern __shared__ int shared_mem[];

	shared_mem[i * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + (j+j_off)];
	shared_mem[(i + block_dim) * block_dim + j] = adj_mat_d[(i + i_off) * comp_V + j + round*block_dim];
	shared_mem[(i + 2*block_dim) * block_dim + j] = adj_mat_d[(i + round * block_dim) * comp_V + (j + j_off)];
    __syncthreads();
	
    #pragma unroll
	for (int k = 0; k < block_dim; k++) {
		if (shared_mem[i * block_dim + j] > shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j])
            shared_mem[i * block_dim + j] = shared_mem[(i + block_dim) * block_dim + k] + shared_mem[(k + 2*block_dim) * block_dim + j];
	}
	
	adj_mat_d[(i + i_off) * comp_V + (j+j_off)] = shared_mem[i * block_dim + j];
	__syncthreads();
}

int main(int argc, char *argv[]){
	/******************************* load data *********************************/
    // only two arguments are allowed
    assert(argc == 3);

    int E, V;
    FILE *in_fp;
    in_fp = fopen(argv[1], "r");
    if(in_fp == NULL) printf("Failed on opening file\n");
    // read in data
    fread(&V, sizeof(int), 1, in_fp);
    fread(&E, sizeof(int), 1, in_fp);

    // compensate V to make V % block_dim == 0
	int comp_V = V + (block_dim - ((V-1) % block_dim + 1));

	//allocate memory
    int *adj_mat; 
    size_t sz = comp_V * comp_V * sizeof(int);
	hipHostMalloc((void**) &adj_mat, sz, hipHostMallocDefault);
	for(int i = 0; i < comp_V; i++){
        for(int j = 0; j < comp_V; j++){
            if(i == j) adj_mat[i*comp_V+j] = 0;
            else adj_mat[i*comp_V+j] = inf;
        }
    }
    // load data to graph
    int src, dst, w;
    while(E--){
        fread(&src, sizeof(int), 1, in_fp);
        fread(&dst, sizeof(int), 1, in_fp);
        fread(&w, sizeof(int), 1, in_fp);
        adj_mat[src*comp_V+dst] = w;
    }
    fclose(in_fp);
    /****************************************************************************/

	int round =  ceil((float) comp_V/block_dim);
    int *adj_mat_d;

    // 2D block
    dim3 threads(block_dim, block_dim);
	
	dim3 p1(1, 1);
	dim3 p2(2, round-1);
    dim3 p3(round, round);
    
    //size_t sz = comp_V * comp_V * sizeof(int);

    hipSetDevice(0);
    // Malloc memory
    hipMalloc((void**) &adj_mat_d, sz);
    hipMemcpy(adj_mat_d, adj_mat, sz, hipMemcpyHostToDevice);

    for(int r = 0; r < round; r++){    
		Phase_1 <<<p1, threads, sizeof(int)*block_dim*block_dim >>>(adj_mat_d, r, block_dim, comp_V);
            
   //     cudaDeviceSynchronize();
			
        Phase_2 <<<p2, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d, r, block_dim, comp_V);
			
   //     cudaDeviceSynchronize();
			
        Phase_3 <<<p3, threads, sizeof(int)*3*block_dim*block_dim >>>(adj_mat_d, r, block_dim, comp_V);
	}
	
	// copy back to host
    hipMemcpy(adj_mat, adj_mat_d, sz, hipMemcpyDeviceToHost);

	// output
    FILE *out_fp;
    out_fp = fopen(argv[2], "wb");
    for(int i = 0; i < V; i++){
        for(int j = 0; j < V; j++){
            fwrite(adj_mat+i*comp_V+j, sizeof(int), 1, out_fp);
        }   
    }   
    fclose(out_fp);

	//free memory
	hipFree(adj_mat_d);
    hipHostFree(adj_mat);
	return 0;
}
