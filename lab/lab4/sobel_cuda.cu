
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8

__device__ int mask[MASK_N][MASK_X][MASK_Y] = { 
    {{ -1, -4, -6, -4, -1},
     { -2, -8,-12, -8, -2},
     {  0,  0,  0,  0,  0}, 
     {  2,  8, 12,  8,  2}, 
     {  1,  4,  6,  4,  1}},
    {{ -1, -2,  0,  2,  1}, 
     { -4, -8,  0,  8,  4}, 
     { -6,-12,  0, 12,  6}, 
     { -4, -8,  0,  8,  4}, 
     { -1, -2,  0,  2,  1}} 
};

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

__global__ void sobel (unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
    int  x, y, i, v, u;
    int  R, G, B;
    double val[MASK_N*3] = {0.0};
    int adjustX, adjustY, xBound, yBound;
    //y = threadIdx.x + blockIdx.x * blockDim.x;
    y = blockIdx.x;
//    for (y = 0; y < height; ++y) {
        for (x = threadIdx.x; x < width; x+=256) {
            for (i = 0; i < MASK_N; ++i) {
                adjustX = (MASK_X % 2) ? 1 : 0;
                adjustY = (MASK_Y % 2) ? 1 : 0;
                xBound = MASK_X /2;
                yBound = MASK_Y /2;

                val[i*3+2] = 0.0;
                val[i*3+1] = 0.0;
                val[i*3] = 0.0;

                for (v = -yBound; v < yBound + adjustY; ++v) {
                    for (u = -xBound; u < xBound + adjustX; ++u) {
                        if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                            R = s[channels * (width * (y+v) + (x+u)) + 2];
                            G = s[channels * (width * (y+v) + (x+u)) + 1];
                            B = s[channels * (width * (y+v) + (x+u)) + 0];
                            val[i*3+2] += R * mask[i][u + xBound][v + yBound];
                            val[i*3+1] += G * mask[i][u + xBound][v + yBound];
                            val[i*3+0] += B * mask[i][u + xBound][v + yBound];
                        }    
                    }
                }
            }

            double totalR = 0.0;
            double totalG = 0.0;
            double totalB = 0.0;
            for (i = 0; i < MASK_N; ++i) {
                totalR += val[i * 3 + 2] * val[i * 3 + 2];
                totalG += val[i * 3 + 1] * val[i * 3 + 1];
                totalB += val[i * 3 + 0] * val[i * 3 + 0];
            }

            totalR = sqrt(totalR) / SCALE;
            totalG = sqrt(totalG) / SCALE;
            totalB = sqrt(totalB) / SCALE;
            const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
            const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
            const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
            t[channels * (width * y + x) + 2] = cR;
            t[channels * (width * y + x) + 1] = cG;
            t[channels * (width * y + x) + 0] = cB;
        }
//    }
}

int main(int argc, char** argv) {

    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char* host_s = NULL;
    read_png(argv[1], &host_s, &height, &width, &channels);
    unsigned char* device_s;
    unsigned char* device_t;
    // malloc memory for GPU
    hipMalloc(&device_s, height * width * channels);
    hipMalloc(&device_t, height * width * channels);

    unsigned char* host_t = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));
    // use cudaMemcpy2D to accelerate
    //cudaMemcpy2D(device_t, channels, device_s, channels, height, width*sizeof(unsigned char), cudaMemcpyHostToDevice);
    hipMemcpy(device_s, host_s, height * width * channels, hipMemcpyHostToDevice);
    // by using multiple threads
    sobel<<<height, 256>>>(device_s, device_t, height, width, channels);
    hipMemcpy(host_t, device_t, height * width * channels, hipMemcpyDeviceToHost);
    write_png(argv[2], host_t, height, width, channels);

    return 0;
}
