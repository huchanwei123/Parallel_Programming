#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

unsigned char *host_s = NULL;       // source image array
unsigned char *host_t = NULL;       // target image array
FILE *fp_s = NULL;                  // source file handler
FILE *fp_t = NULL;                  // target file handler

unsigned int   width, height;       // image width, image height
unsigned int   rgb_raw_data_offset; // RGB raw data offset
unsigned char  bit_per_pixel;       // bit per pixel
unsigned short byte_per_pixel;      // byte per pixel

// bitmap header
unsigned char header[54] = {
	0x42,        // identity : B
	0x4d,        // identity : M
	0, 0, 0, 0,  // file size
	0, 0,        // reserved1
	0, 0,        // reserved2
	54, 0, 0, 0, // RGB data offset
	40, 0, 0, 0, // struct BITMAPINFOHEADER size
	0, 0, 0, 0,  // bmp width
	0, 0, 0, 0,  // bmp height
	1, 0,        // planes
	24, 0,       // bit per pixel
	0, 0, 0, 0,  // compression
	0, 0, 0, 0,  // data size
	0, 0, 0, 0,  // h resolution
	0, 0, 0, 0,  // v resolution 
	0, 0, 0, 0,  // used colors
	0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
// Task 2: Put mask[][][] into Shared Memroy
int mask[MASK_N][MASK_X][MASK_Y] = {
	{{ -1, -4, -6, -4, -1},
	 { -2, -8,-12, -8, -2},
	 {  0,  0,  0,  0,  0},
	 {  2,  8, 12,  8,  2},
	 {  1,  4,  6,  4,  1}},
	{{ -1, -2,  0,  2,  1},
	 { -4, -8,  0,  8,  4},
	 { -6,-12,  0, 12,  6},
	 { -4, -8,  0,  8,  4},
	 { -1, -2,  0,  2,  1}}
};

int read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET); 
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET); 
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// Task 3: Assign host_s to Pinnned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	//host_s = (unsigned char *) malloc((size_t)width * height * byte_per_pixel);
	int err = hipHostMalloc(&host_s,(size_t)width * height * byte_per_pixel, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("malloc images_s error\n");
		return -1;
	}

	// Task 3: Assign host_t to Pinned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	//host_t = (unsigned char *) malloc((size_t) width * height * byte_per_pixel);
	err = hipHostMalloc(&host_t,(size_t)width * height * byte_per_pixel, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("malloc host_t error\n");
		return -1;
	}

	fread(host_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

// declare this as global !!!!
__global__ void sobel (unsigned char *host_s, unsigned char *host_t,
			int *mask_, unsigned int width, unsigned int height,
                        unsigned short byte_per_pixel) 
{
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;

	// Task 2: Put mask[][][] into Shared Memory
	// Hint  : Please declare it in kernel function
	//         Then use some threads to move data from global memory to shared memory
	//         Remember to __syncthreads() after it's done <WHY?>
	// put mask into share !!!!!!!
	__shared__ int mask[MASK_N][MASK_X][MASK_Y];
	int thdIdx_x = threadIdx.x;
	if(thdIdx_x < MASK_X){
		for(int i = 0; i < MASK_N; i++){
			for(int j = 0; j < MASK_Y; j++){
				mask[i][thdIdx_x][j] = mask_[i * MASK_X * MASK_Y + thdIdx_x * MASK_Y + j];
			}
		}
	}		
	__syncthreads();

	// Task 1: Relabel x or y or both into combination of blockIdx, threadIdx ... etc
	// Hint A: We do not have enough threads for each pixels in the image, so what should we do?
	// Hint B: Maybe you can map each y to different threads in different blocks
	for (y = blockIdx.x; y < blockIdx.x+1; ++y) {
		for (x = threadIdx.x; x < width; x+=64) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i * 3 + 2] * val[i * 3 + 2];
				totalG += val[i * 3 + 1] * val[i * 3 + 1];
				totalB += val[i * 3 + 0] * val[i * 3 + 0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			host_t[byte_per_pixel * (width * y + x) + 2] = cR;
			host_t[byte_per_pixel * (width * y + x) + 1] = cG;
			host_t[byte_per_pixel * (width * y + x) + 0] = cB;
		}
	}
}

int write_bmp (const char *fname_t) {
	unsigned int file_size;

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size  
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(host_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int main(int argc, char **argv) {
	// initialize
	hipSetDevice(0);
	
    	assert(argc == 3);
    	const char* input = argv[1];
    	const char* output = argv[2];
	read_bmp(input); // 24 bit gray level image
    

	// Task 1: Allocate memory on GPU
	// Hint  : cudaMalloc ()
	//         What do we need to store on GPU? (input image, output image, ...)
	// declare image array
	unsigned char  *host_s_ = NULL;     
	unsigned char  *host_t_ = NULL;     
	int *mask_ = NULL; 
	hipMalloc((void**)&host_s_, (size_t)width * height * byte_per_pixel);
  	hipMalloc((void**)&host_t_, (size_t)width * height * byte_per_pixel);
	hipMalloc((void**)&mask_, (size_t) sizeof(int) * MASK_N * MASK_Y * MASK_X);

	// Task 1: Memory copy from Host to Device (GPU)
	// Hint  : cudaMemcpy ( ... , cudaMemcpyHostToDevice )
	hipMemcpy(host_s_, host_s, width * height * byte_per_pixel, hipMemcpyHostToDevice);
	hipMemcpy(mask_, mask, sizeof(int) * MASK_N * MASK_Y * MASK_X, hipMemcpyHostToDevice);

	// Task 1: Modify sobel() to CUDA kernel function
	// Hint  : sobel_Kernel <<< ??? , ??? >>> ( ??? );
	sobel <<<height, 64>>> (host_s_, host_t_, mask_, width, height, byte_per_pixel);

	// Task 1: Memory Copy from Device (GPU) to Host
	// Hint  : cudaMemcpy ( ... , cudaMemcpyDeviceToHost )
	hipMemcpy(host_t, host_t_, (size_t)width * height * byte_per_pixel, hipMemcpyDeviceToHost);

	// Task 1: Free memory on device
	// Hint  : cudaFree ( ... )
	hipFree(host_s_);
	hipFree(host_t_);
	hipFree(mask_);
	write_bmp(output);

	// Task 3: Free Pinned memory
	// Hint  : replace free ( ... ) by cudaFreeHost ( ... )
	hipHostFree (host_s);
	hipHostFree (host_t);
}
